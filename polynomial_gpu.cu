// Sourabh Pardeshi - 801081931 - spardes1@uncc.edu
// CUDA Programming


#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>


__global__ void polynomial_expansion (float* poly, int degree, int n, float* array)
{
  //TODO: Write code to use the GPU here!
  //code should write the output back to array
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  
  if (idx < n)
  {
  	float output = 0.0;
  	float xponential = 1.0;
  	
  	for (int i = 0; i <= degree; i++)
  	{
      output += xponential * poly [i];
      xponential *= array [idx];
    }
    
    array [idx] = output;
  }

}


int main (int argc, char* argv[]) {
  //TODO: add usage
  
  if (argc < 3) {
     std::cerr<<"usage: "<<argv[0]<<" n degree"<<std::endl;
     return -1;
  }

  int n = atoi(argv[1]); //TODO: atoi is an unsafe function
  int degree = atoi(argv[2]);
  int nbiter = 1;

  float* array = new float[n];
  float* poly = new float[degree+1];
  for (int i=0; i<n; ++i)
    array[i] = 1.;

  for (int i=0; i<degree+1; ++i)
    poly[i] = 1.;

  
  std::chrono::time_point<std::chrono::system_clock> begin, end;
  begin = std::chrono::system_clock::now();
  
  float *arr_1, *poly_1;
  
  hipMallocManaged (&arr_1, n * sizeof (float));
  hipMallocManaged (&poly_1, (degree + 1) * sizeof (float));
  
  hipMemcpy (arr_1, array, n * sizeof (float), hipMemcpyHostToDevice);
  hipMemcpy (poly_1, poly, n * sizeof (float), hipMemcpyHostToDevice);

  hipDeviceSynchronize();
  end = std::chrono::system_clock::now();
  
  for (int iter = 0; iter<nbiter; ++iter)
    polynomial_expansion <<< (n+255) / 256, 256 >>> (poly_1, degree, n, arr_1);
	
  hipMemcpy(array, arr_1, n * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(arr_1);
  hipFree(poly_1);

  hipDeviceSynchronize();
  end = std::chrono::system_clock::now();
  std::chrono::duration <double> totaltime = (end-begin)/nbiter;

  {
    bool correct = true;
    int ind;
    for (int i=0; i< n; ++i) {
      if (fabs(array[i]-(degree+1))>0.01) {
        correct = false;
	ind = i;
      }
    }
    if (!correct)
      std::cerr<<"Result is incorrect. In particular array["<<ind<<"] should be "<<degree+1<<" not "<< array[ind]<<std::endl;
  }
  

  std::cerr<<array[0]<<std::endl;
  std::cout<<n<<" "<<degree<<" "<<totaltime.count()<<std::endl;

  delete[] array;
  delete[] poly;

  return 0;
}
